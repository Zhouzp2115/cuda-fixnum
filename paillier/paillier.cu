#include "hip/hip_runtime.h"
#include "paillier.h"

#include "fixnum/warp_fixnum.cu"
#include "array/fixnum_array.h"
#include "functions/quorem_preinv.cu"
#include "functions/modexp.cu"
#include "functions/multi_modexp.cu"
#include "modnum/modnum_monty_redc.cu"
#include "modnum/modnum_monty_cios.cu"
#include "functions/paillier_decrypt.cu"
#include "functions/paillier_encrypt.cu"

using namespace std;
using namespace cuFIXNUM;

typedef warp_fixnum<256, u64_fixnum> fixnum;
typedef fixnum_array<fixnum> fixnum_array_;

template <typename fixnum>
struct paillier_encrypt_func
{
    __device__ void operator()(fixnum &z, fixnum n, fixnum m, fixnum r)
    {
        fixnum zz;
        paillier_encrypt<fixnum> enc(n);
        enc(zz, m, r);
        z = zz;
    };
};

template <typename fixnum>
struct paillier_decrypt_func
{
    __device__ void operator()(fixnum &decrypted, fixnum ctx, fixnum lamda, fixnum n, fixnum n_2, fixnum g_lamda_inv)
    {
        fixnum ctx_lamda, res;
        multi_modexp<modnum_monty_redc<fixnum>> mme_n2(n_2);
        quorem_preinv<fixnum> mulmod(n);
        
        //q = L(c^lamda mod n^2)
        mme_n2(ctx_lamda, ctx, lamda);
        fixnum::sub(ctx_lamda ,ctx_lamda ,fixnum::one());
        fixnum q, r;
        mulmod(q, r, fixnum::zero(), ctx_lamda);
        
        //q*g_lamda_inv
        fixnum hi, lo;
        fixnum::mul_wide(hi, lo, q, g_lamda_inv);
        mulmod(res, hi, lo);
        decrypted = res;
    };
};

void paillierSetGPUDevice(int id)
{
    hipSetDevice(id);
}

PublicKey::PublicKey(char *key_file)
{
    FILE *file = fopen(key_file, "rb");
    fread(n, sizeof(uint8_t), 256, file);
    fread(n2, sizeof(uint8_t), 256, file);
    fclose(file);
    
    //init array_n with 10000
    uint8_t *repeated = repeat(n, 10000);
    array_n = (void *)fixnum_array_::create(repeated, 10000 * 256, 256);
    delete[] repeated;

    repeated = repeat(n2, 10000);
    array_n2 = (void *)fixnum_array_::create(repeated, 10000 * 256, 256);
    delete[] repeated;

}

PublicKey::~PublicKey()
{
    fixnum_array_ *ptr = (fixnum_array_ *)array_n;
    delete ptr;
}

uint8_t *PublicKey::repeat(uint8_t *input, int nelts)
{
    uint8_t *repeated = new uint8_t[nelts * 256];
    for (int i = 0; i < nelts; i++)
        memcpy(repeated + i * 256, input, 256);

    return repeated;
}

void *PublicKey::getkeys_array_n(int nelts)
{
    fixnum_array_ *array_n_ptr = (fixnum_array_ *)array_n;

    if (array_n_ptr->length() < nelts)
    {
        delete array_n_ptr;
        uint8_t *repeated = repeat(n, nelts);
        array_n = (void *)fixnum_array_::create(repeated, nelts * 256, 256);

        delete[] repeated;
    }

    return array_n;
}

void *PublicKey::getkeys_array_n2(int nelts)
{
    fixnum_array_ *array_n2_ptr = (fixnum_array_ *)array_n2;

    if (array_n2_ptr->length() < nelts)
    {
        delete array_n2_ptr;
        uint8_t *repeated = repeat(n2, nelts);
        array_n2 = (void *)fixnum_array_::create(repeated, nelts * 256, 256);

        delete[] repeated;
    }

    return array_n2;
}

uint8_t *PublicKey::encrypt(uint8_t *m, uint8_t *r, int nelts, int element_byte_len)
{
    fixnum_array_ *array_m = fixnum_array_::create(m, nelts * element_byte_len, element_byte_len);
    fixnum_array_ *array_r = fixnum_array_::create(r, nelts * element_byte_len, element_byte_len);
    fixnum_array_ *array_n_ptr = (fixnum_array_ *)getkeys_array_n(nelts);

    fixnum_array_ *ctx = fixnum_array_::create(nelts);
    fixnum_array_::template map<paillier_encrypt_func>(ctx, array_n_ptr, array_m, array_r);
    
    uint8_t *ctx_ptr = new uint8_t[nelts * 256];
    int size = nelts;
    ctx->retrieve_all(ctx_ptr, nelts * 256, &size);
    
    delete array_m;
    delete array_r;
 
    return ctx_ptr;
}

PrivateKey::PrivateKey(char *key_file)
{
    FILE *file = fopen(key_file, "rb");
    fread(n, sizeof(uint8_t), 256, file);
    fread(n2, sizeof(uint8_t), 256, file);
    fread(p, sizeof(uint8_t), 256, file);
    fread(q, sizeof(uint8_t), 256, file);
    fread(lamda, sizeof(uint8_t), 256, file);
    fread(lg_inv, sizeof(uint8_t), 256, file);
    fclose(file);
    
    //init  with 10000 element
    uint8_t *repeated = repeat(n, 10000);
    array_n = (void *)fixnum_array_::create(repeated, 10000 * 256, 256);
    delete[] repeated;

    repeated = repeat(n2, 10000);
    array_n2 = (void *)fixnum_array_::create(repeated, 10000 * 256, 256);
    delete[] repeated;

    repeated = repeat(p, 10000);
    array_p = (void *)fixnum_array_::create(repeated, 10000 * 256, 256);
    delete[] repeated;

    repeated = repeat(q, 10000);
    array_q = (void *)fixnum_array_::create(repeated, 10000 * 256, 256);
    delete[] repeated;

    repeated = repeat(lamda, 10000);
    array_lamda = (void *)fixnum_array_::create(repeated, 10000 * 256, 256);
    delete[] repeated;

    repeated = repeat(lg_inv, 10000);
    array_lg_inv = (void *)fixnum_array_::create(repeated, 10000 * 256, 256);
    delete[] repeated;
}

PrivateKey::~PrivateKey()
{
    fixnum_array_ *ptr = (fixnum_array_ *)array_n;
    delete ptr;
    ptr = (fixnum_array_ *)array_n2;
    delete ptr;
    ptr = (fixnum_array_ *)array_p;
    delete ptr;
    ptr = (fixnum_array_ *)array_q;
    delete ptr;
    ptr = (fixnum_array_ *)array_lamda;
    delete ptr;
    ptr = (fixnum_array_ *)array_lg_inv;
    delete ptr;
}

uint8_t *PrivateKey::repeat(uint8_t *input, int nelts)
{
    uint8_t *repeated = new uint8_t[nelts * 256];
    for (int i = 0; i < nelts; i++)
        memcpy(repeated + i * 256, input, 256);

    return repeated;
}

void *PrivateKey::getkeys_array_n(int nelts)
{
    fixnum_array_ *array_n_ptr = (fixnum_array_ *)array_n;

    if (array_n_ptr->length() < nelts)
    {
        delete array_n_ptr;
        uint8_t *repeated = repeat(n, nelts);
        array_n = (void *)fixnum_array_::create(repeated, nelts * 256, 256);

        delete[] repeated;
    }

    return array_n;
}

void *PrivateKey::getkeys_array_n2(int nelts)
{
    fixnum_array_ *array_n2_ptr = (fixnum_array_ *)array_n2;

    if (array_n2_ptr->length() < nelts)
    {
        delete array_n2_ptr;
        uint8_t *repeated = repeat(n2, nelts);
        array_n2 = (void *)fixnum_array_::create(repeated, nelts * 256, 256);

        delete[] repeated;
    }

    return array_n2;
}

void *PrivateKey::getkeys_array_lamda(int nelts)
{
    fixnum_array_ *array_lamda_ptr = (fixnum_array_ *)array_lamda;

    if (array_lamda_ptr->length() < nelts)
    {
        delete array_lamda_ptr;
        uint8_t *repeated = repeat(lamda, nelts);
        array_lamda = (void *)fixnum_array_::create(repeated, nelts * 256, 256);

        delete[] repeated;
    }

    return array_lamda;
}

void *PrivateKey::getkeys_array_lg_nv(int nelts)
{
    fixnum_array_ *array_lg_nv_ptr = (fixnum_array_ *)array_lg_inv;

    if (array_lg_nv_ptr->length() < nelts)
    {
        delete array_lg_nv_ptr;
        uint8_t *repeated = repeat(lg_inv, nelts);
        array_lg_inv = (void *)fixnum_array_::create(repeated, nelts * 256, 256);

        delete[] repeated;
    }

    return array_lg_inv;
}

uint8_t *PrivateKey::decrypt(uint8_t *ctx, int nelts)
{
    fixnum_array_ *array_ctx = fixnum_array_::create(ctx, nelts * 256, 256);
    fixnum_array_ *array_ptx = fixnum_array_::create(nelts);
    fixnum_array_ *array_lamda_ptr = (fixnum_array_ *)getkeys_array_lamda(nelts);
    fixnum_array_ *array_n_ptr = (fixnum_array_ *)getkeys_array_n(nelts);
    fixnum_array_ *array_n2_ptr = (fixnum_array_ *)getkeys_array_n2(nelts);
    fixnum_array_ *array_lg_inv_ptr = (fixnum_array_ *)getkeys_array_lg_nv(nelts);

    fixnum_array_::template map<paillier_decrypt_func>(array_ptx, array_ctx, array_lamda_ptr, array_n_ptr,
                                                       array_n2_ptr, array_lg_inv_ptr);

    uint8_t *ptx_ptr = new uint8_t[nelts * 256];
    int size = nelts;
    array_ptx->retrieve_all(ptx_ptr, nelts * 256, &size);

    delete array_ctx;
    delete array_ptx;

    return ptx_ptr;
}

//efficent
template <typename modnum>
struct my_modexp
{
    typedef typename modnum::fixnum fixnum;

    __device__ void operator()(fixnum &res, fixnum x, fixnum exp, fixnum mod)
    {
        modexp<modnum> me(mod, exp);
        fixnum zz;
        me(zz, x);
        res = zz;
    };
};
template <typename fixnum>
using modexp_cios = my_modexp<modnum_monty_cios<fixnum>>;


template <typename modnum>
struct my_multi_modexp
{
    typedef typename modnum::fixnum fixnum;

    __device__ void operator()(fixnum &z, fixnum x, fixnum e, fixnum mod)
    {
        multi_modexp<modnum> mme(mod);
        fixnum zz;
        mme(zz, x, e);
        z = zz;
    };
};
template <typename fixnum>
using multi_modexp_cios = my_multi_modexp<modnum_monty_cios<fixnum>>;

template <typename modnum>
struct my_mult_mod
{
    typedef typename modnum::fixnum fixnum;

    __device__ void operator()(fixnum &res, fixnum x, fixnum y, fixnum mod)
    {
        quorem_preinv<fixnum> pmod(mod);
        fixnum hi, lo;
        fixnum::mul_wide(hi, lo, x, y);

        fixnum zz;
        pmod(zz, hi, lo);
        res = zz;
    };
};
template <typename fixnum>
using multi_mod = my_mult_mod<modnum_monty_redc<fixnum>>;

/*
uint8_t * dot_he(uint8_t *ctx, uint8_t *ptx, uint8_t *r, uint8_t *r_mask, int batch_size, int input_size, PublicKey *pk)
{
    int bytelen = 256;
    uint8_t *encrypted_r = pk->encrypt(r, r_mask, batch_size, 8);

    fixnum_array_ *array_ctx = fixnum_array_::create(ctx, batch_size * input_size * bytelen, bytelen);
    fixnum_array_ *array_ptx = fixnum_array_::create(ptx, batch_size * input_size * 8, 8);
    fixnum_array_ *array_muled = fixnum_array_::create(batch_size * input_size);
    fixnum_array_ *array_n2 = (fixnum_array_ *)pk->getkeys_array_n2(batch_size * input_size);
    
    //mul
    fixnum_array_::template map<modexp_cios>(array_muled, array_ctx, array_ptx, array_n2);

    uint8_t *muled_ptr = new uint8_t[batch_size * input_size * bytelen];
    int size = batch_size * input_size * bytelen;
    array_muled->retrieve_all(muled_ptr, batch_size * input_size * bytelen, &size);
    
    //add
    fixnum_array_ *array_res = fixnum_array_::create(encrypted_r, batch_size * bytelen, bytelen);
    for (int i = 0; i < input_size; i++)
    {
        fixnum_array_ *array_to_add = fixnum_array_::create(muled_ptr + i * batch_size * bytelen, batch_size * bytelen, bytelen);
        fixnum_array_ *array_added = fixnum_array_::create(batch_size);
        fixnum_array_::template map<multi_mod>(array_added, array_res, array_to_add, array_n2);

        delete array_res;
        delete array_to_add;
        array_res = array_added;
    }

    uint8_t *res_ptr = new uint8_t[batch_size * bytelen];
    size = batch_size;
    array_res->retrieve_all(res_ptr, batch_size * bytelen, &size);

    return res_ptr;
}
*/

uint8_t *sum_func(uint8_t *left, uint8_t *right, fixnum_array_ *array_n2, int nelts)
{
    fixnum_array_ *array_left = fixnum_array_::create(left , nelts * 256, 256);
    fixnum_array_ *array_right = fixnum_array_::create(right , nelts * 256, 256);
    fixnum_array_ *array_res = fixnum_array_::create(nelts);

    fixnum_array_::template map<multi_mod>(array_res, array_left, array_right, array_n2);

    uint8_t *res_ptr = new uint8_t[nelts * 256];
    int size = nelts;
    array_res->retrieve_all(res_ptr, nelts * 256, &size);
    
    delete array_left;
    delete array_right;
    delete array_res;
    return res_ptr;
}

uint8_t *sum(uint8_t *data, fixnum_array_ *array_n2, int nelts, int batch_len)
{
    if (nelts <= 1)
        return data;
    
    if (nelts % 2 == 0)
    {
        nelts = nelts / 2;
        uint8_t *res = sum_func(data, data + nelts * batch_len * 256, array_n2, nelts * batch_len);
        delete[] data;
        return sum(res, array_n2, nelts, batch_len);
    }
    else
    {
        uint8_t *remain = data + (nelts - 1) * batch_len * 256;
        remain = sum_func(remain, data, array_n2, batch_len);
        memcpy(data, remain, batch_len * 256);
        delete[] remain;
        
        nelts = nelts / 2;
        uint8_t *res = sum_func(data, data + nelts * batch_len * 256, array_n2, nelts * batch_len);
        delete[] data;
        return sum(res, array_n2, nelts, batch_len);
    }
    
}

uint8_t * dot_he(uint8_t *ctx, uint8_t *ptx, uint8_t *r, uint8_t *r_mask, int batch_size, int input_size, PublicKey *pk)
{
    int bytelen = 256;
    uint8_t *encrypted_r = pk->encrypt(r, r_mask, batch_size, 8);

    fixnum_array_ *array_ctx = fixnum_array_::create(ctx, batch_size * input_size * bytelen, bytelen);
    fixnum_array_ *array_ptx = fixnum_array_::create(ptx, batch_size * input_size * 8, 8);
    fixnum_array_ *array_muled = fixnum_array_::create(batch_size * input_size);
    fixnum_array_ *array_n2 = (fixnum_array_ *)pk->getkeys_array_n2(batch_size * input_size);
    
    //mul
    fixnum_array_::template map<modexp_cios>(array_muled, array_ctx, array_ptx, array_n2);

    uint8_t *muled_ptr = new uint8_t[batch_size * input_size * bytelen];
    int size = batch_size * input_size * bytelen;
    array_muled->retrieve_all(muled_ptr, batch_size * input_size * bytelen, &size);
    
    //add
    uint8_t *sumed = sum(muled_ptr, array_n2, input_size, batch_size);
    fixnum_array_ *array_r = fixnum_array_::create(encrypted_r, batch_size * bytelen, bytelen);
    fixnum_array_ *array_sumed = fixnum_array_::create(sumed, batch_size * bytelen, bytelen);
    fixnum_array_ *array_res = fixnum_array_::create(batch_size);
    fixnum_array_::template map<multi_mod>(array_res, array_r, array_sumed, array_n2);
    uint8_t *res_ptr = new uint8_t[batch_size * 256];
    size = batch_size;
    array_res->retrieve_all(res_ptr, batch_size * 256, &size);

    delete array_ctx;
    delete array_ptx;
    delete array_muled;
    delete array_r;
    delete array_sumed;
    delete array_res;
    delete encrypted_r;

    return res_ptr;
}