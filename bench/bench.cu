#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstring>
#include <cassert>
#include <unistd.h>

#include "fixnum/warp_fixnum.cu"
#include "array/fixnum_array.h"
#include "functions/quorem_preinv.cu"
#include "functions/modexp.cu"
#include "functions/multi_modexp.cu"
#include "modnum/modnum_monty_redc.cu"
#include "modnum/modnum_monty_cios.cu"
#include "functions/paillier_decrypt.cu"
#include "functions/paillier_encrypt.cu"

using namespace std;
using namespace cuFIXNUM;

/*
k-bit * k-bit = 2k-bit
res = a * b
res - low k bit
*/
template <typename fixnum>
struct mul_lo
{
    __device__ void operator()(fixnum &res, fixnum a, fixnum b)
    {
        fixnum s;
        fixnum::mul_lo(s, a, b);
        res = s;
    }
};

/*
k-bit * k-bit = 2k-bit
res = a * b
res_hi - high k bit
res_lo - low k bit
*/
template <typename fixnum>
struct mul_wide
{
    __device__ void operator()(fixnum &res_hi, fixnum &res_lo, fixnum a, fixnum b)
    {
        fixnum rr, ss;
        fixnum::mul_wide(ss, rr, a, b);
        res_hi = ss;
        res_lo = rr;
    }
};

template <typename fixnum>
struct sqr_wide
{
    __device__ void operator()(fixnum &r, fixnum a)
    {
        fixnum rr, ss;
        fixnum::sqr_wide(ss, rr, a);
        r = ss;
    }
};

/*
modexp<modnum_tp>::modexp(fixnum mod, fixnum exp)
res = x^exp % mod
*/
template <typename modnum>
struct my_modexp
{
    typedef typename modnum::fixnum fixnum;

    __device__ void operator()(fixnum &res, fixnum x, fixnum exp, fixnum mod)
    {
        modexp<modnum> me(mod, exp);
        fixnum zz;
        me(zz, x);
        res = zz;
    };
};

/*
same to modexp   
*/
template <typename modnum>
struct my_multi_modexp
{
    typedef typename modnum::fixnum fixnum;

    __device__ void operator()(fixnum &z, fixnum x, fixnum e, fixnum mod)
    {
        multi_modexp<modnum> mme(mod);
        fixnum zz;
        mme(zz, x, e);
        z = zz;
    };
};

template <typename modnum>
struct my_mult_mod
{
    typedef typename modnum::fixnum fixnum;

    __device__ void operator()(fixnum &res, fixnum x, fixnum y, fixnum mod)
    {
        quorem_preinv<fixnum> pmod(mod);
        fixnum hi, lo;
        fixnum::mul_wide(hi, lo, x, y);

        fixnum zz;
        pmod(zz, hi, lo);
        res = zz;
    };
};

uint8_t *long2bytes(long *input, int length, int fn_bytes)
{
    uint8_t *bytes = new uint8_t[length * fn_bytes];
    memset(bytes, 0x00, length * fn_bytes);
    for (int i = 0; i < length; i++)
    {
        uint8_t *ptr = (uint8_t *)(&input[i]);
        bytes[(i + 1) * fn_bytes - 1] = ptr[0];
        bytes[(i + 1) * fn_bytes - 2] = ptr[1];
        bytes[(i + 1) * fn_bytes - 3] = ptr[2];
        bytes[(i + 1) * fn_bytes - 4] = ptr[3];
    }

    return bytes;
}

long *bytes2long(uint8_t *bytes, int length, int fn_bytes = 256)
{
    long *res = new long[length / fn_bytes];
    for (int i = 0; i < length / fn_bytes; i++)
    {
        uint8_t *ptr = (uint8_t *)&res[i];
        ptr[0] = bytes[(i + 1) * fn_bytes - 1];
        ptr[1] = bytes[(i + 1) * fn_bytes - 2];
        ptr[2] = bytes[(i + 1) * fn_bytes - 3];
        ptr[3] = bytes[(i + 1) * fn_bytes - 4];
    }

    return res;
}

void printBytes(uint8_t *bytes, int length, int fn_bytes = 256)
{
    for (int i = 0; i < length; i++)
    {
        printf("%x ", bytes[i]);
        if ((i + 1) % fn_bytes == 0)
            printf("\n");
    }
}

template <int fn_bytes, typename word_fixnum, template <typename> class Func>
void bench(int nelts)
{
    typedef warp_fixnum<fn_bytes, word_fixnum> fixnum;
    typedef fixnum_array<fixnum> fixnum_array;

    if (nelts == 0)
    {
        puts(" -*-  nelts == 0; skipping...  -*-");
        return;
    }

    uint8_t *input_0 = new uint8_t[fn_bytes * nelts];
    uint8_t *input_1 = new uint8_t[fn_bytes * nelts];
    uint8_t *input_2 = new uint8_t[fn_bytes * nelts];
    uint8_t *data = new uint8_t[fn_bytes * nelts];
    memset(input_0, 0x00, fn_bytes * nelts);
    memset(input_1, 0x00, fn_bytes * nelts);
    memset(input_2, 0x00, fn_bytes * nelts);
    memset(data, 0x00, fn_bytes * nelts);
    for (int i = 0; i < fn_bytes * nelts; i++)
    {
        input_0[i] = 0x11;
        input_1[i] = 0x12;
        input_2[i] = 0x13;
    }

    /*
    for (int i = 0; i < nelts * fn_bytes; i++)
    {
        input_0[i] = 0x01;
        input_1[i] = 0x01;
        input_2[i] = 0x02;
    }
    */

    fixnum_array *res_hi, *res_lo, *in_0, *in_1, *in_2;
    in_0 = fixnum_array::create(input_0, fn_bytes * nelts, fn_bytes);
    in_1 = fixnum_array::create(input_1, fn_bytes * nelts, fn_bytes);
    in_2 = fixnum_array::create(input_2, fn_bytes * nelts, fn_bytes);
    res_hi = fixnum_array::create(nelts);
    res_lo = fixnum_array::create(nelts);

    // warm up
    //fixnum_array::template map<Func>(res_lo, in_0, in_0, in_1);

    clock_t c = clock();
    fixnum_array::template map<Func>(res_lo, in_0, in_1, in_2);
    c = clock() - c;

    //printf("res lo\n");
    res_lo->retrieve_all(data, nelts * fn_bytes, &nelts);
    //printBytes(data, nelts * fn_bytes, fn_bytes);
    //printf("\n");

    double secinv = (double)CLOCKS_PER_SEC / c;
    double total_MiB = fixnum::BYTES * (double)nelts / (1 << 20);
    printf(" %4d   %3d    %6.1f   %7.3f  %12.1f\n",
           fixnum::BITS, fixnum::digit::BITS, total_MiB,
           1 / secinv, nelts * 1e-3 * secinv);

    delete in_0;
    delete in_1;
    delete in_2;
    delete res_hi;
    delete res_lo;
    delete[] input_0;
    delete[] input_1;
    delete[] input_2;
    delete[] data;
}

template <template <typename> class Func>
void bench_func(const char *fn_name, int nelts)
{
    printf("Function: %s, #elts: %de3\n", fn_name, (int)(nelts * 1e-3));
    printf("fixnum digit  total data   time       Kops/s\n");
    printf(" bits  bits     (MiB)    (seconds)\n");
    bench<4, u32_fixnum, Func>(nelts);
    bench<8, u32_fixnum, Func>(nelts);
    bench<16, u32_fixnum, Func>(nelts);
    bench<32, u32_fixnum, Func>(nelts);
    bench<64, u32_fixnum, Func>(nelts);
    bench<128, u32_fixnum, Func>(nelts);
    puts("");

    bench<8, u64_fixnum, Func>(nelts);
    bench<16, u64_fixnum, Func>(nelts);
    bench<32, u64_fixnum, Func>(nelts);
    bench<64, u64_fixnum, Func>(nelts);
    bench<128, u64_fixnum, Func>(nelts);
    bench<256, u64_fixnum, Func>(nelts);
    puts("");
}

template <typename fixnum>
using modexp_redc = my_modexp<modnum_monty_redc<fixnum>>;

template <typename fixnum>
using modexp_cios = my_modexp<modnum_monty_cios<fixnum>>;

template <typename fixnum>
using multi_modexp_redc = my_multi_modexp<modnum_monty_redc<fixnum>>;

template <typename fixnum>
using multi_modexp_cios = my_multi_modexp<modnum_monty_cios<fixnum>>;

template <typename fixnum>
using multi_mod_redc = my_mult_mod<modnum_monty_redc<fixnum>>;

template <typename fixnum>
using multi_mod_cios = my_mult_mod<modnum_monty_cios<fixnum>>;

/*
template< typename fixnum >
struct pencrypt {
    __device__ void operator()(fixnum &z, fixnum p, fixnum q, fixnum r, fixnum m) {
        fixnum n, zz;
        fixnum::mul_lo(n, p, q);
        paillier_encrypt<fixnum> enc(n);
        enc(zz, m, r);
        z = zz;
    };
};
*/

template <typename fixnum>
struct pencrypt
{
    __device__ void operator()(fixnum &z, fixnum n, fixnum m, fixnum r)
    {
        fixnum zz;
        paillier_encrypt<fixnum> enc(n);
        enc(zz, m, r);
        z = zz;
    };
};

template <typename fixnum>
struct paillier_encrypt_manner
{
    __device__ void operator()(fixnum &encrypted, fixnum n, fixnum n_2, fixnum g, fixnum m, fixnum r)
    {
        fixnum g_m, r_n, res;
        multi_modexp<modnum_monty_redc<fixnum>> mme(n_2);
        quorem_preinv<fixnum> mulmod(n_2);
        mme(g_m, g, m);
        mme(r_n, r, n);
        
        fixnum hi, lo;
        fixnum::mul_wide(hi, lo, g_m, r_n);
        mulmod(res, hi, lo);
        encrypted = res;
    };
};

/*
template< typename fixnum >
struct pdecrypt {
    __device__ void operator()(fixnum &z, fixnum ct, fixnum p, fixnum q, fixnum r, fixnum m) {
        
        if (fixnum::cmp(p, q) == 0
              || fixnum::cmp(r, p) == 0
              || fixnum::cmp(r, q) == 0) {
            printf("equal \n");
            z = fixnum::zero();
            return;
        }
        
        paillier_decrypt<fixnum> dec(p, q);
        fixnum n, zz;
        dec(zz, fixnum::zero(), ct);
        //dec(z, fixnum::zero(), ct);
        //fixnum::mul_lo(n, p, q);
        //quorem_preinv<fixnum> qr(n);
        //qr(m, fixnum::zero(), m);

        // z = (z != m)
        //z = fixnum::digit( !! fixnum::cmp(zz, m));
        z = zz;
    };
};
*/

template <typename fixnum>
struct pdecrypt
{
    __device__ void operator()(fixnum &z, fixnum ct, fixnum p, fixnum q)
    {
        paillier_decrypt<fixnum> dec(p, q);
        fixnum zz;
        dec(zz, fixnum::zero(), ct);
        z = zz;
    };
};

template <typename fixnum>
struct paillier_decrypt_manner
{
    __device__ void operator()(fixnum &decrypted, fixnum ctx, fixnum lamda, fixnum n, fixnum n_2, fixnum g_lamda_inv)
    {
        fixnum ctx_lamda, res;
        multi_modexp<modnum_monty_redc<fixnum>> mme_n2(n_2);
        quorem_preinv<fixnum> mulmod(n);
        
        //q = L(c^lamda mod n^2)
        mme_n2(ctx_lamda, ctx, lamda);
        fixnum::sub(ctx_lamda ,ctx_lamda ,fixnum::one());
        fixnum q, r;
        mulmod(q, r, fixnum::zero(), ctx_lamda);
        
        //q*g_lamda_inv
        fixnum hi, lo;
        fixnum::mul_wide(hi, lo, q, g_lamda_inv);
        mulmod(res, hi, lo);
        decrypted = res;
    };
};

void host_function(int num)
{

    // fixnum represents 256-byte numbers, using a 64-bit "basic fixnum".
    typedef warp_fixnum<256, u64_fixnum> fixnum;
    typedef fixnum_array<fixnum> fixnum_array;

    int nelts = num, byteslen = 256;
    fixnum_array *ct, *pt, *p, *q, *n, *g, *n2, *lamda, *g_lamda_inv;

    uint8_t *private_key_p = new uint8_t[nelts * byteslen];
    uint8_t *private_key_q = new uint8_t[nelts * byteslen];
    uint8_t *private_key_lamda = new uint8_t[nelts * byteslen];
    uint8_t *private_key_g_lamda_inv = new uint8_t[nelts * byteslen];
    uint8_t *public_key_n = new uint8_t[nelts * byteslen];
    uint8_t *public_key_g = new uint8_t[nelts * byteslen];
    uint8_t *public_key_n2 = new uint8_t[nelts * byteslen];
    uint8_t *data = new uint8_t[nelts * byteslen];
    memset(private_key_p, 0x00, nelts * byteslen);
    memset(private_key_q, 0x00, nelts * byteslen);
    memset(private_key_lamda, 0x00, nelts * byteslen);
    memset(private_key_g_lamda_inv, 0x00, nelts * byteslen);
    memset(public_key_n, 0x00, nelts * byteslen);
    memset(public_key_g, 0x00, nelts * byteslen);
    memset(public_key_n2, 0x00, nelts * byteslen);
    memset(data, 0x00, nelts * byteslen);

    for (int i = 0; i < nelts; i++)
    {
        private_key_p[i * byteslen] = 0x05;
        private_key_q[i * byteslen] = 0x07;
        private_key_lamda[i * byteslen] = 0x0c;
        private_key_g_lamda_inv[i * byteslen] = 0x03;

        public_key_n[i * byteslen] = 0x23;
        public_key_g[i * byteslen] = 0x24;
        public_key_n2[i * byteslen] = 0xc9;
        public_key_n2[i * byteslen + 1] = 0x04;
    }

    for (int i = 0; i < nelts * byteslen; i++)
    {
        private_key_p[i] = 0x05;
        private_key_q[i] = 0x07;
        private_key_lamda[i] = 0x0c;
        private_key_g_lamda_inv[i] = 0x03;

        public_key_n[i] = 0x23;
        public_key_g[i] = 0x24;
        public_key_n2[i] = 0xc9;
    }

    p = fixnum_array::create(private_key_p, nelts * byteslen, byteslen);
    q = fixnum_array::create(private_key_q, nelts * byteslen, byteslen);
    n = fixnum_array::create(public_key_n, nelts * byteslen, byteslen);
    g = fixnum_array::create(public_key_g, nelts * byteslen, byteslen);
    n2 = fixnum_array::create(public_key_n2, nelts * byteslen, byteslen);
    lamda = fixnum_array::create(private_key_lamda, nelts * byteslen, byteslen);
    g_lamda_inv = fixnum_array::create(private_key_g_lamda_inv, nelts * byteslen, byteslen);

    ct = fixnum_array::create(nelts);
    pt = fixnum_array::create(nelts);
    printf("p->length() = %d \n", p->length());
    printf("q->length() = %d \n", q->length());
    printf("n->length() = %d \n", n->length());
    printf("g->length() = %d \n", g->length());
    printf("n2->length() = %d \n", n2->length());
    printf("lamda->length() = %d \n", lamda->length());
    printf("g_lamda_inv->length() = %d \n", g_lamda_inv->length());
    printf("ct->length() = %d \n", ct->length());
    printf("pt->length() = %d \n", pt->length());

    for (int i = 0; i < nelts; i++)
    {
        data[i * byteslen] = 0x03;
    }

    for (int i = 0; i < nelts * byteslen; i++)
    {
        data[i] = 0x11;
    }


    fixnum_array *m = fixnum_array::create(data, nelts * byteslen, byteslen);
    fixnum_array *r = fixnum_array::create(data, nelts * byteslen, byteslen);
    printf("m->length() = %d \n", m->length());
    printf("r->length() = %d \n", r->length());

    //warm up
    //fixnum_array::template map<paillier_encrypt_manner>(ct, n, n2, g, m, r);

    clock_t t = clock();
    //fixnum_array::template map<paillier_encrypt_manner>(ct, n, n2, g, m, r);
    t = clock() - t;
    double secinv = (double)CLOCKS_PER_SEC / t;
    printf("manner encrypt time cost:%f \n", 1 / secinv);

    t = clock();
    fixnum_array::template map<pencrypt>(ct, n, m, r);
    t = clock() - t;
    secinv = (double)CLOCKS_PER_SEC / t;
    printf("encrypt time cost:%f \n", 1 / secinv);

    //warm up
    //fixnum_array::template map<paillier_decrypt_manner>(pt, ct, lamda, n, n2, g_lamda_inv);

    t = clock();
    fixnum_array::template map<paillier_decrypt_manner>(pt, ct, lamda, n, n2, g_lamda_inv);
    t = clock() - t;
    secinv = (double)CLOCKS_PER_SEC / t;
    printf("manner decrypt time cost:%f \n", 1 / secinv);

    uint8_t *res = new uint8_t[nelts * fixnum::BYTES];
    memset(res, 0x00, nelts * fixnum::BYTES);
    pt->retrieve_all(res, nelts * fixnum::BYTES, &nelts);
    printBytes(res, 1, fixnum::BYTES);

    t = clock();
    //fixnum_array::template map<pdecrypt>(pt, ct, p, q);
    t = clock() - t;
    secinv = (double)CLOCKS_PER_SEC / t;
    printf("decrypt time cost:%f \n", 1 / secinv);

    /*
    int len = 1000;
    printf("p \n");
    memset(data, 0x00, nelts * fixnum::BYTES);
    p->retrieve_all(data, nelts * fixnum::BYTES, &len);
    printBytes(data, nelts * fixnum::BYTES);
    printf("q \n");
    memset(data, 0x00, nelts * fixnum::BYTES);
    q->retrieve_all(data, nelts * fixnum::BYTES, &len);
    printBytes(data, nelts * fixnum::BYTES);
    printf("m \n");
    memset(data, 0x00, nelts * fixnum::BYTES);
    m->retrieve_all(data, nelts * fixnum::BYTES, &len);
    printBytes(data, nelts * fixnum::BYTES);
    printf("r \n");
    memset(data, 0x00, nelts * fixnum::BYTES);
    r->retrieve_all(data, nelts * fixnum::BYTES, &len);
    printBytes(data, nelts * fixnum::BYTES);

    printf("ct \n");
    memset(data, 0x00, nelts * fixnum::BYTES);
    ct->retrieve_all(data, nelts * fixnum::BYTES, &len);
    printBytes(data, nelts * fixnum::BYTES);

    printf("pt \n");
    memset(data, 0x00, nelts * fixnum::BYTES);
    pt->retrieve_all(data, nelts * fixnum::BYTES, &len);
    printBytes(data, nelts * fixnum::BYTES);
    */

    delete[] private_key_p;
    delete[] private_key_q;
    delete[] public_key_n;
    delete[] data;
    delete ct;
    delete pt;
    delete p;
    delete q;
}

int main(int argc, char *argv[])
{
    long m = 1;
    if (argc > 1)
        m = atol(argv[1]);
    m = std::max(m, 1000L);
    
    /*
    bench_func<mul_lo>("mul_lo", m);
    puts("");
    bench_func<mul_wide>("mul_wide", m);
    puts("");
    bench_func<sqr_wide>("sqr_wide", m);
    puts("");
    bench_func<modexp_redc>("modexp redc", m);
    puts("");
    bench_func<modexp_cios>("modexp cios", m);
    puts("");
    bench_func<multi_modexp_redc>("multi modexp redc", m);
    puts("");
    */
    
    bench_func<modexp_cios>("multi modexp cios", m);
    puts("");
    bench_func<modexp_redc>("multi modexp redc", m);
    puts("");
    
    bench_func<multi_mod_redc>("mult mod redc", m);
    puts("");
    bench_func<multi_mod_cios>("mult mod cios", m);
    puts("");

    return 0;
}
